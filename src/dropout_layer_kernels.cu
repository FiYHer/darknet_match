#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>
#include <cstring>

#include "dropout_layer.h"
#include "dark_cuda.h"
#include "utils.h"
#include "blas.h"


__global__ void dropblock_fast_kernel(float *rand, float prob, int w, int h, int spatial, int filters, int block_size, float *drop_blocks_scale, float *output)
{
    const int threads = BLOCK;
    const int id = threadIdx.x;
    const int f = blockIdx.x % filters;
    const int b = blockIdx.x / filters;

    __shared__ int prob_block;
    __shared__ int index_block;

    if (id == 0) {
        prob_block = 1.0 * 1000000;
        index_block = -1;
    }
    __syncthreads();

    int i;
    for (i = id; i < spatial; i += threads) {
        int index = b*spatial*f + f*spatial + i;

        if (rand[index] < prob) {
            //Chose with the lowest rand[i]
            int new_val = rand[index] * 1000000;
            int old_val = atomicMin(&prob_block, new_val);
            if (new_val < old_val) {
                index_block = i;
                //if (b == 0) printf("\n rand[i] = %f, prob = %f, b = %d, f = %d, i = %d, index_block = %d \n", rand[i], prob, b, f, i, index_block);
            }
        }

    }
    __syncthreads();
    if (index_block == -1) return;


    int b_x = index_block % w;
    int b_y = index_block / w;

    b_x = max(0, min(b_x, w - block_size));
    b_y = max(0, min(b_y, h - block_size));

    int block_square_size = block_size * block_size;

    for (i = id; i < block_square_size; i += threads)
    {
        int i_x = i % w;
        int i_y = i / w;

        int x = b_x + i_x;
        int y = b_y + i_y;

        if (x < w && y < h) {
            int index = b*spatial*f + f*spatial + y*w + x;

            output[index] = 0;
        }
    }

    if (id == 0 && drop_blocks_scale) {
        atomicAdd(&drop_blocks_scale[b], 1);
        //if(b == 0) printf("\n index_block = %d \n", index_block);
    }

}

__global__ void set_scales_dropblock_kernel(float *drop_blocks_scale, int block_size_w, int block_size_h, int outputs, int batch)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index >= batch) return;

    const float prob = drop_blocks_scale[index] * block_size_w * block_size_h / (float)outputs;
    const float scale = 1.0f / (1.0f - prob);
    drop_blocks_scale[index] = scale;
}

__global__ void scale_dropblock_kernel(float *output, int size, int outputs, float *drop_blocks_scale)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index >= size) return;

    const int b = index / outputs;
    output[index] *= drop_blocks_scale[b];
}


__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id < size) input[id] = (rand[id] < prob) ? 0 : input[id]*scale;
}


void forward_dropout_layer_gpu(dropout_layer l, network_state state)
{
    if (!state.train) return;
    int iteration_num = get_current_iteration(state.net); // (*state.net.seen) / (state.net.batch*state.net.subdivisions);
    //if (iteration_num < state.net.burn_in) return;

    // We gradually increase the block size and the probability of dropout - during the first half of the training
    float multiplier = 1.0;
    if(iteration_num < (state.net.max_batches*0.85))
        multiplier = (iteration_num / (float)(state.net.max_batches*0.85));

    // dropblock
    if (l.dropblock) {
        //l.probability = 1 / keep_prob
        //const int max_blocks_per_channel = 10;
        const float cur_prob = l.probability * multiplier;
        const float cur_scale = 1.f / (1.f - cur_prob);

        int block_width = l.dropblock_size_abs *multiplier;
        int block_height = l.dropblock_size_abs *multiplier;

        if (l.dropblock_size_rel) {
            block_width = l.dropblock_size_rel * l.w * multiplier;
            block_height = l.dropblock_size_rel * l.h * multiplier;
        }

        block_width = max_val_cmp(1, block_width);
        block_height = max_val_cmp(1, block_height);

        block_width = min_val_cmp(l.w, block_width);
        block_height = min_val_cmp(l.h, block_height);

        const int block_size = min_val_cmp(block_width, block_height);
        const float block_prob = cur_prob / (block_size*block_size);

        int size = l.inputs*l.batch;
        cuda_random(l.rand_gpu, size);

        fill_ongpu(l.batch, 0, l.drop_blocks_scale_gpu, 1);

        int num_blocks = l.batch * l.c;
        dropblock_fast_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> > (l.rand_gpu, block_prob, l.w, l.h, l.w*l.h, l.c, block_size, l.drop_blocks_scale_gpu, state.input);
        CHECK_CUDA(hipPeekAtLastError());

        num_blocks = get_number_of_blocks(l.batch, BLOCK);
        set_scales_dropblock_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> > (l.drop_blocks_scale_gpu, block_size, block_size, l.outputs, l.batch);
        CHECK_CUDA(hipPeekAtLastError());

        /*
        cuda_pull_array(l.drop_blocks_scale_gpu, l.drop_blocks_scale, l.batch);

        for (int b = 0; b < l.batch; ++b) {
            const float prob = l.drop_blocks_scale[b] * block_size * block_size / (float)l.outputs;
            const float scale = 1.0f / (1.0f - prob);
            //printf(" %d x %d - block_size = %d, block_size*block_size = %d , ", l.w, l.h, block_size, block_size*block_size);
            //printf(" , l.drop_blocks_scale[b] = %f, prob = %f, calc scale = %f \t cur_prob = %f, cur_scale = %f \n",
            //    l.drop_blocks_scale[b], prob, scale, cur_prob, cur_scale);
            l.drop_blocks_scale[b] = scale;
        }

        cuda_push_array(l.drop_blocks_scale_gpu, l.drop_blocks_scale, l.batch);
        */

        num_blocks = get_number_of_blocks(l.outputs * l.batch, BLOCK);
        scale_dropblock_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> > (state.input, l.outputs * l.batch, l.outputs, l.drop_blocks_scale_gpu);
        //scal_ongpu(l.outputs * l.batch, cur_scale, state.input, 1);
        //scal_ongpu(l.outputs * l.batch, l.drop_blocks_scale[0], state.input, 1);
        CHECK_CUDA(hipPeekAtLastError());

    }
    // dropout
    else {
        int size = l.inputs*l.batch;
        cuda_random(l.rand_gpu, size);
        /*
        int i;
        for(i = 0; i < size; ++i){
            layer.rand[i] = rand_uniform();
        }
        cuda_push_array(layer.rand_gpu, layer.rand, size);
        */

        yoloswag420blazeit360noscope << <cuda_gridsize(size), BLOCK, 0, get_cuda_stream() >> > (state.input, size, l.rand_gpu, l.probability, l.scale);
        CHECK_CUDA(hipPeekAtLastError());
    }
}

void backward_dropout_layer_gpu(dropout_layer l, network_state state)
{
    if(!state.delta) return;
    //int iteration_num = get_current_iteration(state.net); //(*state.net.seen) / (state.net.batch*state.net.subdivisions);
    //if (iteration_num < state.net.burn_in) return;

    int size = l.inputs*l.batch;

    // dropblock
    if (l.dropblock) {
        int iteration_num = get_current_iteration(state.net); //(*state.net.seen) / (state.net.batch*state.net.subdivisions);
        float multiplier = 1.0;
        if (iteration_num < (state.net.max_batches*0.85))
            multiplier = (iteration_num / (float)(state.net.max_batches*0.85));

        const float cur_prob = l.probability * multiplier;
        const float cur_scale = 1.f / (1.f - cur_prob);

        int block_width = l.dropblock_size_abs * multiplier;
        int block_height = l.dropblock_size_abs * multiplier;

        if (l.dropblock_size_rel) {
            block_width = l.dropblock_size_rel * l.w * multiplier;
            block_height = l.dropblock_size_rel * l.h * multiplier;
        }

        block_width = max_val_cmp(1, block_width);
        block_height = max_val_cmp(1, block_height);

        block_width = min_val_cmp(l.w, block_width);
        block_height = min_val_cmp(l.h, block_height);

        const int block_size = min_val_cmp(block_width, block_height);
        const float block_prob = cur_prob / (block_size*block_size);

        int num_blocks = l.batch * l.c;
        dropblock_fast_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> > (l.rand_gpu, block_prob, l.w, l.h, l.w*l.h, l.c, block_size, NULL, state.delta);
        CHECK_CUDA(hipPeekAtLastError());

        num_blocks = get_number_of_blocks(l.outputs * l.batch, BLOCK);
        scale_dropblock_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> > (state.delta, l.outputs * l.batch, l.outputs, l.drop_blocks_scale_gpu);
        //scal_ongpu(l.outputs * l.batch, cur_scale, state.input, 1);
        //scal_ongpu(l.outputs * l.batch, l.drop_blocks_scale[0], state.input, 1);
        CHECK_CUDA(hipPeekAtLastError());

        //drop_block_kernel << <cuda_gridsize(size), BLOCK, 0, get_cuda_stream() >> > (state.delta, size, l.rand_gpu, l.scale);
        //CHECK_CUDA(hipPeekAtLastError());
    }
    // dropout
    else {
        yoloswag420blazeit360noscope << <cuda_gridsize(size), BLOCK, 0, get_cuda_stream() >> > (state.delta, size, l.rand_gpu, l.probability, l.scale);
        CHECK_CUDA(hipPeekAtLastError());
    }
}
